#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#include "lib/doctest/doctest.hpp"
#include "lib/nanobench/nanobench.h"
#include "primitives/matrix.hpp"
#include "random_utils.hpp"
#include "test/test_utils.hpp"
#include "primitives/color.hpp"

void handleCudaError (hipError_t e, const char* identifier, bool exits = true) {

    fprintf(stdout, "Checking for error in %s... ", identifier);

	if (e == hipSuccess) {
        fprintf(stdout, "No Error Found!\n");
        return;
    }

	fprintf(stdout, "%s %s: %s\n",
		identifier,
		hipGetErrorName(e),
		hipGetErrorString(e)
	);

	fflush(NULL);

	if (exits) exit(1);
}

#define HEIGHT 256
#define WIDTH 256

#define SIZE sizeof(Matrix<HEIGHT, WIDTH>)

__global__ void add (Matrix<HEIGHT, WIDTH>* sum, Matrix<HEIGHT, WIDTH>* lhs, Matrix<HEIGHT, WIDTH>* rhs) {

    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.y) + threadIdx.x;

    (*sum)[row][col] = (*lhs)[row][col] + (*rhs)[row][col];
}

TEST_CASE("Compare Addition") {

    Matrix<HEIGHT, WIDTH> host_lhs;
    Matrix<HEIGHT, WIDTH> host_rhs;

    for (int row = 0; row < HEIGHT; ++row) {
        for (int col = 0; col < WIDTH; ++col) {
            host_lhs[row][col] = randomDouble();
            host_rhs[row][col] = randomDouble();
        }
    }

    Matrix<HEIGHT, WIDTH> host_sum;
    Matrix<HEIGHT, WIDTH> empty_host_sum;

    // Device Memory

    Matrix<HEIGHT, WIDTH>* device_lhs;
    Matrix<HEIGHT, WIDTH>* device_rhs;

    hipMalloc(&device_lhs, SIZE);
    hipMemcpy(device_lhs, &host_lhs, SIZE, hipMemcpyHostToDevice);

    hipMalloc(&device_rhs, SIZE);
    hipMemcpy(device_rhs, &host_rhs, SIZE, hipMemcpyHostToDevice);

    Matrix<HEIGHT, WIDTH>* device_sum;

    hipMalloc(&device_sum, SIZE);
    hipMemcpy(device_sum, &host_sum, SIZE, hipMemcpyHostToDevice);

    dim3 grid_dims(8, 8);
    dim3 block_dims(32, 32);

    // Benchmark

    ankerl::nanobench::Bench b;

    /**

    b.minEpochIterations(1000).run("Serial + Operation", [&] {

        host_sum = host_lhs + host_rhs;

    });

    */
    
    host_sum = host_lhs + host_rhs;

    /**

    b.minEpochIterations(1000).run("Parallel + Operation", [&] {

        add<<<grid_dims, block_dims>>>(device_sum, device_lhs, device_rhs);
        hipDeviceSynchronize();

    });

    */

    add<<<grid_dims, block_dims>>>(device_sum, device_lhs, device_rhs);
    hipDeviceSynchronize();

    hipMemcpy(&empty_host_sum, device_sum, SIZE, hipMemcpyDeviceToHost);

    CHECK_MATRIX(host_sum, empty_host_sum);
}

#define PIXEL_HEIGHT 4
#define PIXEL_WIDTH 4
#define PIXEL_DEPTH 4
#define MULTIPLIER 2

__global__ void multiply (Color* pixels, double multiplier) {

    int row = blockIdx.y;
    int col = blockIdx.x;
    int depth = threadIdx.x;

    pixels[(depth * PIXEL_HEIGHT * PIXEL_WIDTH) + (row * PIXEL_WIDTH) + col] *= 2;
}

TEST_CASE("Testing Unified (Managed) Memory") {

    Color* pixels;
    hipError_t e;

    const int height = PIXEL_HEIGHT;
    const int width = PIXEL_WIDTH;
    const int depth = PIXEL_DEPTH;
    const double multiplier = MULTIPLIER;

    size_t size = sizeof(Color) * height * width * depth;

    e = hipMallocManaged(&pixels, size);
    handleCudaError(e, "Managed Malloc");

    Color* expected_result = (Color*)(malloc(size));

    for (int row = 0; row < height; ++row) {
        for (int col = 0; col < width; ++col) {
            for (int level = 0; level < depth; ++level) {

                Color val { fabs(randomDouble()), fabs(randomDouble()), fabs(randomDouble()) };
                pixels[(level * height * width) + (row * width) + col] = val;
                std::cout << pixels[(level * height * width) + (row * width) + col] << "\n\n";
                expected_result[(level * height * width) + (row * width) + col] = val * multiplier;

            }
        }
    }

    ankerl::nanobench::Bench b;

    /**

    b.epochIterations(1).run("Pixel 3D Array Multiply", [&] {

        dim3 grid_dims(height, width);
        dim3 block_dims(depth);

        multiply<<<grid_dims, block_dims>>>(pixels, multiplier);

        e = hipDeviceSynchronize();
        handleCudaError(e, "Synchronize");
    });

    */

    dim3 grid_dims(height, width);
    dim3 block_dims(depth);

    multiply<<<grid_dims, block_dims>>>(pixels, multiplier);

    e = hipDeviceSynchronize();
    handleCudaError(e, "Synchronize");

    for (int row = 0; row < height; ++row) {
        for (int col = 0; col < width; ++col) {
            for (int level = 0; level < depth; ++level) {
                Color actual_pixel = pixels[(level * height * width) + (row * width) + col];
                Color expected_pixel = expected_result[(level * height * width) + (row * width) + col];
                std::cout << actual_pixel << "\n\n";
                CHECK_COLOR(actual_pixel, expected_pixel);
            }
        }
    }
}

/**
__global__ void pitched_array_test (hipPitchedPtr pitched_ptr) {

    double* elements = (double*)(pitched_ptr.ptr);
    size_t pitch = pitched_ptr.pitch;
    int width = pitched_ptr.xsize;
    int height = pitched_ptr.ysize;

    int row = blockIdx.y;
    int col = blockIdx.x;
    int depth = threadIdx.x;

    double* element = elements + ((depth * height * pitch) + (row * pitch) + col);

    *element *= 2;
}
 *

TEST_CASE("Testing cudaPitchPtr") {

    const int width = 8;
    const int height = 8;
    const int depth = 8;

    double* host_nums = (double*) malloc(sizeof(double) * width * height * depth);
    double* host_result = (double*) malloc(sizeof(double) * width * height * depth);
    double* extra_host_result = (double*) malloc(sizeof(double) * width * height * depth);

    for (int row = 0; row < height; ++row) {
        for (int col = 0; col < width; ++col) {
            for (int level = 0; level < depth; ++level) {
                int i = level + row + col;
                host_nums[(level * height * width) + (row * width) + col] = i;
                host_result[(level * height * width) + (row * width) + col] = i * 2;
            }
        }
    }

    hipArray *pixels;

    hipMalloc3DArray(&pixels, );


    //hipExtent pixelExtent = make_hipExtent(sizeof(Color) * width, height, depth);

    //hipMalloc3D(&pixels, pixelExtent);

    hipMemcpy3DParms params;

    params.srcPtr = make_hipPitchedPtr(pixels, sizeof(Color) * width, width, height);


}

*/
