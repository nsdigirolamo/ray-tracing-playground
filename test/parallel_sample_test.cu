#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#include "lib/doctest/doctest.hpp"
#include "primitives/matrix.hpp"
#include "random_utils.hpp"
#include "test/test_utils.hpp"
#include "primitives/color.hpp"

#define HEIGHT 256
#define WIDTH 256

#define SIZE sizeof(Matrix<HEIGHT, WIDTH>)

__global__ void add (Matrix<HEIGHT, WIDTH>* sum, Matrix<HEIGHT, WIDTH>* lhs, Matrix<HEIGHT, WIDTH>* rhs) {

    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.y) + threadIdx.x;

    (*sum)[row][col] = (*lhs)[row][col] + (*rhs)[row][col];
}

TEST_CASE("Compare Addition") {

    Matrix<HEIGHT, WIDTH> host_lhs;
    Matrix<HEIGHT, WIDTH> host_rhs;

    for (int row = 0; row < HEIGHT; ++row) {
        for (int col = 0; col < WIDTH; ++col) {
            host_lhs[row][col] = randomDouble();
            host_rhs[row][col] = randomDouble();
        }
    }

    Matrix<HEIGHT, WIDTH> host_sum;
    Matrix<HEIGHT, WIDTH> empty_host_sum;

    Matrix<HEIGHT, WIDTH>* device_lhs;
    Matrix<HEIGHT, WIDTH>* device_rhs;

    hipMalloc(&device_lhs, SIZE);
    hipMemcpy(device_lhs, &host_lhs, SIZE, hipMemcpyHostToDevice);

    hipMalloc(&device_rhs, SIZE);
    hipMemcpy(device_rhs, &host_rhs, SIZE, hipMemcpyHostToDevice);

    Matrix<HEIGHT, WIDTH>* device_sum;

    hipMalloc(&device_sum, SIZE);
    hipMemcpy(device_sum, &host_sum, SIZE, hipMemcpyHostToDevice);

    dim3 grid_dims(8, 8);
    dim3 block_dims(32, 32);
    
    host_sum = host_lhs + host_rhs;

    add<<<grid_dims, block_dims>>>(device_sum, device_lhs, device_rhs);
    hipDeviceSynchronize();

    hipMemcpy(&empty_host_sum, device_sum, SIZE, hipMemcpyDeviceToHost);

    CHECK_MATRIX(host_sum, empty_host_sum);
}

#define PIXEL_HEIGHT 4
#define PIXEL_WIDTH 4
#define PIXEL_DEPTH 4
#define MULTIPLIER 2

__global__ void multiply (Color* pixels) {

    int row = blockIdx.y;
    int col = blockIdx.x;
    int depth = threadIdx.x;

    pixels[(depth * PIXEL_HEIGHT * PIXEL_WIDTH) + (row * PIXEL_WIDTH) + col] *= MULTIPLIER;
}

TEST_CASE("Testing Unified (Managed) Memory") {

    Color* pixels;
    size_t size = sizeof(Color) * PIXEL_HEIGHT * PIXEL_WIDTH * PIXEL_DEPTH;

    hipMallocManaged(&pixels, size);

    Color* expected_result = (Color*)(malloc(size));

    for (int row = 0; row < PIXEL_HEIGHT; ++row) {
        for (int col = 0; col < PIXEL_WIDTH; ++col) {
            for (int level = 0; level < PIXEL_DEPTH; ++level) {

                Color val { fabs(randomDouble()), fabs(randomDouble()), fabs(randomDouble()) };
                pixels[(level * PIXEL_HEIGHT * PIXEL_WIDTH) + (row * PIXEL_WIDTH) + col] = val;
                expected_result[(level * PIXEL_HEIGHT * PIXEL_WIDTH) + (row * PIXEL_WIDTH) + col] = val * MULTIPLIER;

            }
        }
    }

    dim3 grid_dims(PIXEL_HEIGHT, PIXEL_WIDTH);
    dim3 block_dims(PIXEL_DEPTH);

    multiply<<<grid_dims, block_dims>>>(pixels);
    hipDeviceSynchronize();

    for (int row = 0; row < PIXEL_HEIGHT; ++row) {
        for (int col = 0; col < PIXEL_WIDTH; ++col) {
            for (int level = 0; level < PIXEL_DEPTH; ++level) {
                Color actual_pixel = pixels[(level * PIXEL_HEIGHT * PIXEL_WIDTH) + (row * PIXEL_WIDTH) + col];
                Color expected_pixel = expected_result[(level * PIXEL_HEIGHT * PIXEL_WIDTH) + (row * PIXEL_WIDTH) + col];
                CHECK_COLOR(actual_pixel, expected_pixel);
            }
        }
    }
}