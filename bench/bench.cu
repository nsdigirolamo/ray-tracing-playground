#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <hip/hip_runtime.h>
#include <iostream>

#include "lib/doctest/doctest.hpp"
#include "lib/nanobench/nanobench.h"
#include "primitives/matrix.hpp"
#include "random_utils.hpp"
#include "test/test_utils.hpp"

#define HEIGHT 256
#define WIDTH 256

#define SIZE sizeof(Matrix<HEIGHT, WIDTH>)

__global__ void add (Matrix<HEIGHT, WIDTH>* sum, Matrix<HEIGHT, WIDTH>* lhs, Matrix<HEIGHT, WIDTH>* rhs) {

    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.y) + threadIdx.x;

    (*sum)[row][col] = (*lhs)[row][col] + (*rhs)[row][col];
}

TEST_CASE("Compare Addition") {

    Matrix<HEIGHT, WIDTH> host_lhs;
    Matrix<HEIGHT, WIDTH> host_rhs;

    for (int row = 0; row < HEIGHT; ++row) {
        for (int col = 0; col < WIDTH; ++col) {
            host_lhs[row][col] = randomDouble();
            host_rhs[row][col] = randomDouble();
        }
    }

    Matrix<HEIGHT, WIDTH> host_sum;
    Matrix<HEIGHT, WIDTH> empty_host_sum;

    // Device Memory

    Matrix<HEIGHT, WIDTH>* device_lhs;
    Matrix<HEIGHT, WIDTH>* device_rhs;

    hipMalloc(&device_lhs, SIZE);
    hipMemcpy(device_lhs, &host_lhs, SIZE, hipMemcpyHostToDevice);

    hipMalloc(&device_rhs, SIZE);
    hipMemcpy(device_rhs, &host_rhs, SIZE, hipMemcpyHostToDevice);

    Matrix<HEIGHT, WIDTH>* device_sum;

    hipMalloc(&device_sum, SIZE);
    hipMemcpy(device_sum, &host_sum, SIZE, hipMemcpyHostToDevice);

    dim3 grid_dims(8, 8);
    dim3 block_dims(32, 32);

    // Benchmark

    ankerl::nanobench::Bench b;

    b.minEpochIterations(1000).run("Serial + Operation", [&] {

        host_sum = host_lhs + host_rhs;

    });

    b.minEpochIterations(1000).run("Parallel + Operation", [&] {

        add<<<grid_dims, block_dims>>>(device_sum, device_lhs, device_rhs);
        hipDeviceSynchronize();

    });

    hipMemcpy(&empty_host_sum, device_sum, SIZE, hipMemcpyDeviceToHost);

    CHECK_MATRIX(host_sum, empty_host_sum);
}
